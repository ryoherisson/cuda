#include <hip/hip_runtime.h>
#include <stdio.h>

#define CHECK(call)                                                         \
{                                                                           \
    const hipError_t error = call;                                         \
    if (error != hipSuccess)                                               \
    {                                                                       \
        printf("Error: %s:%d, ", __FILE__, __LINE__);                       \
        printf("code:%d, reason:%s\n", error, hipGetErrorString(error));   \
        exit(1);                                                            \
    }                                                                       \
}

void printMatrix(int *C, const int nx, const int ny) {
    int *ic = C;
    printf("\nMatrix: (%d.%d)\n", nx, ny);
    for (int iy = 0; iy < ny; iy++) {
        for (int ix = 0; ix < nx; ix++) {
            printf("%3d", ic[ix]);
        }
        ic += nx;
        printf("\n");
    }
    printf("\n");
    return;
}

__global__ void printThreadIndex(int *A, const int nx, const int ny) {
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;

    printf("thread_id (%d,%d) block_id (%d,%d) coordinate (%d,%d) "
           "global index %2d ival %2d\n", threadIdx.x, threadIdx.y,
                                          blockIdx.x, blockIdx.y,
                                          ix, iy, idx, A[idx]);
}

int main(int argc, char **argv) {
    printf("%s Starting...\n", argv[0]);

    // setup device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // matrix setting
    int nx = 8;
    int ny = 6;
    int nxy = nx * ny;
    int nBytes = nxy * sizeof(float);

    // malloc host memory
    int *h_A;
    h_A = (int *)malloc(nBytes);

    // initialize host matrix with integer
    for (int i = 0; i < nxy; i++) {
        h_A[i] = i;
    }
    printMatrix(h_A, nx, ny);

    // malloc device memory
    int *d_MatA;
    CHECK(hipMalloc((void **)&d_MatA, nBytes));

    // transfer data from host to device
    CHECK(hipMemcpy(d_MatA, h_A, nBytes, hipMemcpyHostToDevice));

    // set up execution configuration
    dim3 block(4, 2);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

    // invole the kernel
    printThreadIndex<<< grid, block >>>(d_MatA, nx, ny);
    CHECK(hipDeviceSynchronize());

    // free host and device memory
    CHECK(hipFree(d_MatA));
    free(h_A);

    // reset device
    CHECK(hipDeviceReset());

    return (0);
}