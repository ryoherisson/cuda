#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void print_threadIdx()
{
    // printf("threasdIdx.x: %d, threadIdx.y: %d, threadIdx.z: %d\n", threadIdx.x, threadIdx.y, threadIdx.z);
    printf("blockIdx.x: %d, blockIdx.y: %d, blockIdx.z: %d, blockDim.x: %d, blockDim.y: %d, gridDim.x: %d, gridDim.y: %d\n",
        blockIdx.x, blockIdx.y, blockIdx.z, blockDim.x, blockDim.y, gridDim.x, gridDim.y);
}

int main()
{
    int nx, ny;
    nx = 16;
    ny = 16;
    dim3 block(8, 8);
    dim3 grid(nx / block.x, ny / block.y);

    print_threadIdx<<<grid, block>>>();
    hipDeviceSynchronize();

    hipDeviceReset();

    return 0;
}