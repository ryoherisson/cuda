#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void unique_gid_calculation_2d(int *data) {
  int tid = threadIdx.x;
  int block_offset = blockIdx.x * blockDim.x;
  int row_offset = gridDim.x * blockDim.x * blockIdx.y;
  int gid = tid + block_offset + row_offset;
  printf("blockIdx.x: %d, blockIdx.y: %d, threadIdx.x: %d, gid: %d, - data: "
         "%d \n",
         blockIdx.x, blockIdx.y, tid, gid, data[gid]);
}

int main() {
  int array_size = 16;
  int array_byte_size = sizeof(int) * array_size;
  int h_data[] = {23, 9, 4, 53, 65, 12, 1, 33, 87, 45, 23, 12, 342, 56, 44, 99};

  for (int i = 0; i < array_size; i++) {
    printf("%d ", h_data[i]);
  }
  printf("\n \n");

  int *d_data;
  hipMalloc((void **)&d_data, array_byte_size);
  hipMemcpy(d_data, h_data, array_byte_size, hipMemcpyHostToDevice);

  dim3 block(4);
  dim3 grid(2, 2);

  unique_gid_calculation_2d<<<grid, block>>>(d_data);
  hipDeviceSynchronize();

  hipDeviceReset();

  return 0;
}